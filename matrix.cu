#include<iostream>
#include<hip/hip_runtime.h>
using namespace std;

__global__ void mulmax(int *A,int *B,int *C,int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < n && col < n)
    {
        int sum = 0;
        for (int i = 0; i < n; i++)
        {
            //sum += A[row * n + i] * B[i * n + col];
            sum += A[row * n + i] * B[col + n * i];
        }
        C[row * n + col] = sum;
    }
}

void initialize(int* matrix, int n)
{
    for (int i = 0; i < n*n; i++)
    {
        matrix[i] = rand() % 10;
    }
}

void print(int* matrix, int n)
{
    for (int row = 0; row < n; row++)
    {
        for (int col = 0; col < n; col++)
        {
            cout << matrix[row * n + col] << " ";
        }
        cout << '\n';
    }
    cout << '\n';
}

int main()
{
    int n=4;
    int *A,*B,*C;
    int size=n*n*sizeof(int);

    hipHostMalloc(&A,size, hipHostMallocDefault);
    hipHostMalloc(&B,size, hipHostMallocDefault);
    hipHostMalloc(&C,size, hipHostMallocDefault);

    initialize(A, n);
    initialize(B, n);

    cout << "Matrix A: "<<endl;
    print(A, n);
    cout << "Matrix B: "<<endl;
    print(B, n);

    int *da,*db,*dc;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);

    hipMemcpy(da,A,size,hipMemcpyHostToDevice);
    hipMemcpy(db,B,size,hipMemcpyHostToDevice);

    int thread=n*n;
    dim3 dimBlock(thread, thread); // Block dimensions (16x16 threads per block)
    dim3 numBlocks((n + dimBlock.x - 1) / dimBlock.x, (n + dimBlock.y - 1) / dimBlock.y); // Grid dimensions

    mulmax<<<numBlocks,dimBlock>>>(da,db,dc,n);

    hipMemcpy(C,dc,size,hipMemcpyDeviceToHost);

    cout << "Addition: "<<endl;
    print(C, n);

    hipFree(da);
    hipFree(db);
    hipFree(dc);

    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(C);

    return 0;
}