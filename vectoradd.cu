#include<iostream>
#include<hip/hip_runtime.h>

__global__ void addvect(int *A,int *B,int *C,int n)
{
    int i= blockIdx.x *  blockDim.x + threadIdx.x;

    if(i<n)
    {
        C[i]=A[i]+B[i];
    }
}

int main()
{
    int n=100;
    int *A,*B,*C;
    int size=n*sizeof(int);

    hipHostMalloc(&A,size, hipHostMallocDefault);
    hipHostMalloc(&B,size, hipHostMallocDefault);
    hipHostMalloc(&C,size, hipHostMallocDefault);

    for(int i=0;i<n;i++)
    {
        A[i]=i;
        B[i]=i*2;
    }

    int *da,*db,*dc;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);

    hipMemcpy(da,A,size,hipMemcpyHostToDevice);
    hipMemcpy(db,B,size,hipMemcpyHostToDevice);

    int blocksize=256;
    int numblock=(n+blocksize-1)/blocksize;

    addvect<<<numblock,blocksize>>>(da,db,dc,n);

    hipMemcpy(C,dc,size,hipMemcpyDeviceToHost);

    for(int i=0;i<10;i++)
    {
        printf("%d + %d = %d \n",A[i],B[i],C[i]);
    }

    hipFree(da);
    hipFree(db);
    hipFree(dc);

    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(C);

    return 0;
}